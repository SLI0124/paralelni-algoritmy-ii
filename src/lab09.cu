#include "hip/hip_runtime.h"
#include <cudaDefs.h>
#include <time.h>
#include <math.h>
#include <benchmark.h>

namespace lab09 {
	constexpr unsigned int N = 1 << 22;
	constexpr unsigned int MEMSIZE = N * sizeof(unsigned int);
	constexpr unsigned int NO_LOOPS = 100;
	constexpr unsigned int TPB = 256;
	constexpr unsigned int GRID_SIZE = (N + TPB - 1) / TPB;

	constexpr unsigned int NO_TEST_PHASES = 10;

	void fillData(unsigned int* data, const unsigned int length)
	{
		for (unsigned int i = 0; i < length; i++)
		{
			data[i] = 1;
		}
	}

	void printData(const unsigned int* data, const unsigned int length)
	{
		if (data == 0) return;
		for (unsigned int i = 0; i < length; i++)
		{
			printf("%u ", data[i]);
		}
	}


	__global__ void kernel(const unsigned int* a, const unsigned int* b, const unsigned int length, unsigned int* c)
	{
		const unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
		//TODO:  thread block loop
		if (tid < length)
		{
			c[tid] = a[tid] + b[tid];
		}
	}

	////////////////////////////////////////////////////////////////////////////////////////////////////
	/// <summary>	Tests 1. - single stream, async calling </summary>
	////////////////////////////////////////////////////////////////////////////////////////////////////
	void test1()
	{
		unsigned int* a, * b, * c;
		unsigned int* da, * db, * dc;

		// paged-locked allocation
		hipHostAlloc((void**)&a, NO_LOOPS * MEMSIZE, hipHostMallocDefault);
		hipHostAlloc((void**)&b, NO_LOOPS * MEMSIZE, hipHostMallocDefault);
		hipHostAlloc((void**)&c, NO_LOOPS * MEMSIZE, hipHostMallocDefault);

		fillData(a, NO_LOOPS * N);
		fillData(b, NO_LOOPS * N);

		// Data chunks on GPU
		hipMalloc((void**)&da, MEMSIZE);
		hipMalloc((void**)&db, MEMSIZE);
		hipMalloc((void**)&dc, MEMSIZE);

		//TODO: create stream
		hipStream_t stream;
		hipStreamCreate(&stream);


		auto lambda = [&]()
			{
				unsigned int dataOffset = 0;
				for (int i = 0; i < NO_LOOPS; i++)
				{
					//TODO:  copy a->da, b->db
					hipMemcpyAsync(da, &a[dataOffset], MEMSIZE, hipMemcpyHostToDevice, stream);
					hipMemcpyAsync(db, &b[dataOffset], MEMSIZE, hipMemcpyHostToDevice, stream);
					//TODO:  run the kernel in the stream
					kernel << <GRID_SIZE, TPB, 0, stream >> > (da, db, N, dc);
					//TODO:  copy dc->c
					hipMemcpyAsync(&c[dataOffset], dc, MEMSIZE, hipMemcpyDeviceToHost, stream);

					dataOffset += N;
				}
			};
		float gpuTime = GPUTIME(NO_TEST_PHASES, lambda());

		hipStreamSynchronize(stream); // wait for stream to finish
		hipStreamDestroy(stream);
		hipDeviceSynchronize();
		printf("\x1B[93m[GPU time] %s: %f ms\033[0m\n", __PRETTY_FUNCTION__, gpuTime);

		//printData(c, 100);

		hipFree(da);
		hipFree(db);
		hipFree(dc);

		hipHostFree(a);
		hipHostFree(b);
		hipHostFree(c);
	}

	////////////////////////////////////////////////////////////////////////////////////////////////////
	/// <summary>	Tests 2. - two streams - depth first approach </summary>
	////////////////////////////////////////////////////////////////////////////////////////////////////
	void test2()
	{
		//TODO: reuse the source code of above mentioned method test1()
		unsigned int* a, * b, * c;
		unsigned int* da0, * db0, * dc0;
		unsigned int* da1, * db1, * dc1;

		// paged-locked allocation
		hipHostAlloc((void**)&a, NO_LOOPS * MEMSIZE, hipHostMallocDefault);
		hipHostAlloc((void**)&b, NO_LOOPS * MEMSIZE, hipHostMallocDefault);
		hipHostAlloc((void**)&c, NO_LOOPS * MEMSIZE, hipHostMallocDefault);

		fillData(a, NO_LOOPS * N);
		fillData(b, NO_LOOPS * N);

		// Data chunks on GPU
		hipMalloc((void**)&da0, MEMSIZE);
		hipMalloc((void**)&db0, MEMSIZE);
		hipMalloc((void**)&dc0, MEMSIZE);

		hipMalloc((void**)&da1, MEMSIZE);
		hipMalloc((void**)&db1, MEMSIZE);
		hipMalloc((void**)&dc1, MEMSIZE);

		//TODO: create stream
		hipStream_t stream0;
		hipStreamCreate(&stream0);
		hipStream_t stream1;
		hipStreamCreate(&stream1);


		auto lambda = [&]()
			{
				unsigned int dataOffset = 0;
				for (int i = 0; i < NO_LOOPS; i += 2)
				{
					//TODO:  copy a->da, b->db
					hipMemcpyAsync(da0, &a[dataOffset], MEMSIZE, hipMemcpyHostToDevice, stream0);
					hipMemcpyAsync(db0, &b[dataOffset], MEMSIZE, hipMemcpyHostToDevice, stream0);
					//TODO:  run the kernel in the stream
					kernel << <GRID_SIZE, TPB, 0, stream0 >> > (da0, db0, N, dc0);
					//TODO:  copy dc->c
					hipMemcpyAsync(&c[dataOffset], dc0, MEMSIZE, hipMemcpyDeviceToHost, stream0);
					dataOffset += N;

					//TODO:  copy a->da, b->db
					hipMemcpyAsync(da1, &a[dataOffset], MEMSIZE, hipMemcpyHostToDevice, stream1);
					hipMemcpyAsync(db1, &b[dataOffset], MEMSIZE, hipMemcpyHostToDevice, stream1);
					//TODO:  run the kernel in the stream
					kernel << <GRID_SIZE, TPB, 0, stream1 >> > (da1, db1, N, dc1);
					//TODO:  copy dc->c
					hipMemcpyAsync(&c[dataOffset], dc1, MEMSIZE, hipMemcpyDeviceToHost, stream1);
					dataOffset += N;
				}
			};
		float gpuTime = GPUTIME(NO_TEST_PHASES, lambda());

		hipStreamSynchronize(stream0); // wait for stream to finish
		hipStreamSynchronize(stream1); // wait for stream to finish
		hipStreamDestroy(stream0);
		hipStreamDestroy(stream1);
		hipDeviceSynchronize();
		printf("\x1B[93m[GPU time] %s: %f ms\033[0m\n", __PRETTY_FUNCTION__, gpuTime);

		//printData(c, 100);

		hipFree(da0);
		hipFree(db0);
		hipFree(dc0);

		hipFree(da1);
		hipFree(db1);
		hipFree(dc1);

		hipHostFree(a);
		hipHostFree(b);
		hipHostFree(c);

	}

	////////////////////////////////////////////////////////////////////////////////////////////////////
	/// <summary>	Tests 3. - two streams - breadth first approach</summary>
	////////////////////////////////////////////////////////////////////////////////////////////////////
	void test3()
	{
		//TODO: reuse the source code of above mentioned method test1()
		unsigned int* a, * b, * c;
		unsigned int* da0, * db0, * dc0;
		unsigned int* da1, * db1, * dc1;

		// paged-locked allocation
		hipHostAlloc((void**)&a, NO_LOOPS * MEMSIZE, hipHostMallocDefault);
		hipHostAlloc((void**)&b, NO_LOOPS * MEMSIZE, hipHostMallocDefault);
		hipHostAlloc((void**)&c, NO_LOOPS * MEMSIZE, hipHostMallocDefault);

		fillData(a, NO_LOOPS * N);
		fillData(b, NO_LOOPS * N);

		// Data chunks on GPU
		hipMalloc((void**)&da0, MEMSIZE);
		hipMalloc((void**)&db0, MEMSIZE);
		hipMalloc((void**)&dc0, MEMSIZE);

		hipMalloc((void**)&da1, MEMSIZE);
		hipMalloc((void**)&db1, MEMSIZE);
		hipMalloc((void**)&dc1, MEMSIZE);

		//TODO: create stream
		hipStream_t stream0;
		hipStreamCreate(&stream0);
		hipStream_t stream1;
		hipStreamCreate(&stream1);


		auto lambda = [&]()
			{
				unsigned int dataOffset0 = 0;
				unsigned int dataOffset1 = N;

				for (int i = 0; i < NO_LOOPS; i += 2)
				{
					//TODO:  copy a->da, b->db
					hipMemcpyAsync(da0, &a[dataOffset0], MEMSIZE, hipMemcpyHostToDevice, stream0);
					hipMemcpyAsync(da1, &a[dataOffset1], MEMSIZE, hipMemcpyHostToDevice, stream1);
					hipMemcpyAsync(db0, &b[dataOffset0], MEMSIZE, hipMemcpyHostToDevice, stream0);
					hipMemcpyAsync(db1, &b[dataOffset1], MEMSIZE, hipMemcpyHostToDevice, stream1);
					//TODO:  run the kernel in the stream
					kernel << <GRID_SIZE, TPB, 0, stream0 >> > (da0, db0, N, dc0);
					kernel << <GRID_SIZE, TPB, 0, stream1 >> > (da1, db1, N, dc1);
					//TODO:  copy dc->c
					hipMemcpyAsync(&c[dataOffset0], dc0, MEMSIZE, hipMemcpyDeviceToHost, stream0);
					hipMemcpyAsync(&c[dataOffset1], dc1, MEMSIZE, hipMemcpyDeviceToHost, stream1);

					dataOffset0 += 2 * N;
					dataOffset1 += 2 * N;
				}
			};
		float gpuTime = GPUTIME(NO_TEST_PHASES, lambda());

		hipStreamSynchronize(stream0); // wait for stream to finish
		hipStreamSynchronize(stream1); // wait for stream to finish
		hipStreamDestroy(stream0);
		hipStreamDestroy(stream1);
		hipDeviceSynchronize();
		printf("\x1B[93m[GPU time] %s: %f ms\033[0m\n", __PRETTY_FUNCTION__, gpuTime);

		//printData(c, 100);

		hipFree(da0);
		hipFree(db0);
		hipFree(dc0);

		hipFree(da1);
		hipFree(db1);
		hipFree(dc1);

		hipHostFree(a);
		hipHostFree(b);
		hipHostFree(c);
	}

	void run()
	{
		// run this in release mode and run it with Ctrl+F5
		test1();
		test2();
		test3();
	}
} // namespace lab09