#include "hip/hip_runtime.h"
#include <cudaDefs.h>
#include <hipblas.h>

namespace lab10
{
	hipblasStatus_t status = hipblasStatus_t();
	hipblasHandle_t handle = hipblasHandle_t();

	const unsigned int N = 5;
	const unsigned int dim = 3;
	const unsigned int MEMSIZE = N * dim * sizeof(float);
	const unsigned int THREAD_PER_BLOCK = 128;
	const unsigned int GRID_SIZE = (N * dim + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK;

	void fillData(float* data, const unsigned int length, const unsigned int dim)
	{
		unsigned int id = 0;
		for (unsigned int i = 0; i < length; i++)
		{
			for (unsigned int j = 0; j < dim; j++)
			{
				data[id++] = i & 255;   //=i%256
			}
		}
	}

	void fillDataWithNumber(float* data, const unsigned int length, const unsigned int dim, const float number)
	{
		unsigned int id = 0;
		for (unsigned int i = 0; i < length; i++)
		{
			for (unsigned int j = 0; j < dim; j++)
			{
				data[id++] = number;
			}
		}
	}

	__global__ void kernelPowerTwo(const float* a, const float* b, const unsigned int length, float* a2, float* b2)
	{
		//TODO:
		int idx = blockIdx.x * blockDim.x + threadIdx.x;
		int jump = blockDim.x * gridDim.x;

		while (idx < length)
		{
			float tmp = a[idx];
			a2[idx] = tmp * tmp;
			tmp = b[idx];
			b2[idx] = tmp * tmp;
			idx += jump;
		}
	}

	void run()
	{
		status = hipblasCreate(&handle);

		float alpha, beta;
		float* a, * b, * m;
		float* da, * da2, * db, * db2, * dm;
		float* ones, * dones;

		// paged-locked allocation
		hipHostAlloc((void**)&a, MEMSIZE, hipHostMallocDefault);
		hipHostAlloc((void**)&b, MEMSIZE, hipHostMallocDefault);
		hipHostAlloc((void**)&ones, MEMSIZE, hipHostMallocDefault);
		hipHostAlloc((void**)&m, N * N * sizeof(float), hipHostMallocDefault);

		hipMalloc((void**)&da, MEMSIZE);
		hipMalloc((void**)&da2, MEMSIZE);
		hipMalloc((void**)&db, MEMSIZE);
		hipMalloc((void**)&db2, MEMSIZE);
		hipMalloc((void**)&dones, MEMSIZE);
		hipMalloc((void**)&dm, N * N * sizeof(float));

		fillData(a, N, dim);
		fillData(b, N, dim);
		fillDataWithNumber(ones, N, dim, 1.0f);

		//Copy data to DEVICE
		hipMemcpy(da, a, MEMSIZE, hipMemcpyHostToDevice);
		hipMemcpy(db, b, MEMSIZE, hipMemcpyHostToDevice);
		hipMemcpy(dones, ones, MEMSIZE, hipMemcpyHostToDevice);

		//TODO 1: Process a -> a^2  and b->b^2
		kernelPowerTwo << <GRID_SIZE, THREAD_PER_BLOCK >> > (da, db, N * dim, da2, db2);

		//TODO 2: Process a^2 + b^2 using CUBLAS //pair-wise operation such that the result is dm[N*N] matrix
		// T => Transpose
		// N => do absolutly nothing

		// k = dim
		// lda = leading dimension of A, skip to next row, dim
		// second leading dimension of B, skip to next row, B
		alpha = 1.0f;
		beta = 0.0f;
		hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, N, N, dim, &alpha, da2, dim, dones, dim, &beta, dm, N);
		checkDeviceMatrix<float>(dm, sizeof(float) * N, N, N, "%f ", "A^2 + M => M");

		alpha = 1.0f;
		beta = 1.0f;
		hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, N, N, dim, &alpha, dones, dim, db2, dim, &beta, dm, N);
		checkDeviceMatrix<float>(dm, sizeof(float) * N, N, N, "%f ", "B^2 + M => M");

		//TODO 3: Process -2ab and sum with previous result stored in dm using CUBLAS
		checkDeviceMatrix<float>(dm, sizeof(float) * N, N, N, "%f ", "-2AB + M => M");
		alpha = -2.0f;
		//beta = 1.0f;
		hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, N, N, dim, &alpha, da, dim, db, dim, &beta, dm, N);

		checkDeviceMatrix<float>(da, sizeof(float) * dim, N, dim, "%f ", "A");
		checkDeviceMatrix<float>(da2, sizeof(float) * dim, N, dim, "%f ", "A^2");
		checkDeviceMatrix<float>(db, sizeof(float) * dim, N, dim, "%f ", "B");
		checkDeviceMatrix<float>(db2, sizeof(float) * dim, N, dim, "%f ", "B^2");
		checkDeviceMatrix<float>(dones, sizeof(float) * dim, N, dim, "%f ", "ONES");
		checkDeviceMatrix<float>(dm, sizeof(float) * N, N, N, "%f ", "M");

		hipFree(da);
		hipFree(da2);
		hipFree(db);
		hipFree(db2);
		hipFree(dm);
		hipFree(dones);
		hipHostFree(a);
		hipHostFree(b);
		hipHostFree(m);
		hipHostFree(ones);

		status = hipblasDestroy(handle);
	}

} // namespace lab10