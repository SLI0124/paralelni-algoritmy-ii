#include "hip/hip_runtime.h"
// includes, cudaimageWidth
#include <cudaDefs.h>

#include <hip/hip_vector_types.h>			// normalize method

#include <benchmark.h>
#include <imageManager.h>
#include <imageUtils.cuh>

namespace lab05 {

#define TPB_1D 8						// ThreadsPerBlock in one dimension
#define TPB_2D TPB_1D*TPB_1D			// ThreadsPerBlock = TPB_1D*TPB_1D (2D block)

	using namespace gpubenchmark;
	using DT = float;


	__host__ TextureInfo createTextureObjectFrom2DArray(const ImageInfo<DT>& ii)
	{
		TextureInfo ti;

		// Size info
		ti.size = { ii.width, ii.height, 1 };

		//Texture Data settings
		//ti.texChannelDesc = ... hipCreateChannelDesc<DT>();  // hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindUnsigned);
		ti.texChannelDesc = hipCreateChannelDesc<DT>();  // hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindUnsigned);  
		//checkCudaErrors(hipMallocArray(...));
		checkCudaErrors(hipMallocArray(&ti.texArrayData, &ti.texChannelDesc, ii.width, ii.height));
		//checkCudaErrors(hipMemcpyToArray(...));
		checkCudaErrors(hipMemcpyToArray(ti.texArrayData, 0, 0, ii.dPtr, ii.pitch * ii.height, hipMemcpyDeviceToDevice));

		// Specify texture resource
		//ti.resDesc.resType = ... 
		ti.resDesc.resType = hipResourceTypeArray;
		//ti.resDesc.res.array.array = ... 
		ti.resDesc.res.array.array = ti.texArrayData;

		// Specify texture object parameters
		//ti.texDesc.addressMode[0] = ...
		ti.texDesc.addressMode[0] = hipAddressModeClamp;
		//ti.texDesc.addressMode[1] = ... 
		ti.texDesc.addressMode[1] = hipAddressModeClamp;
		//ti.texDesc.filterMode = ... 
		ti.texDesc.filterMode = hipFilterModePoint;
		//ti.texDesc.readMode = ... 
		ti.texDesc.readMode = hipReadModeElementType;
		//ti.texDesc.normalizedCoords = false;
		ti.texDesc.normalizedCoords = false;

		// Create texture object
		//checkCudaErrors(hipCreateTextureObject(...));
		checkCudaErrors(hipCreateTextureObject(&ti.texObj, &ti.resDesc, &ti.texDesc, nullptr));

		return ti;
	}

	__global__ void texKernel(const hipTextureObject_t srcTex, const unsigned int srcWidth, const unsigned int srcHeight, float* dst)
	{
		// TODO
		int tx = (int)(blockIdx.x * blockDim.x + threadIdx.x);
		int ty = (int)(blockIdx.y * blockDim.y + threadIdx.y);

		if ((tx >= srcWidth) || (ty >= srcHeight)) return;

		dst[ty * srcWidth + tx] = tex2D<float>(srcTex, tx, ty);
	}


	void run()
	{
		FreeImage_Initialise();

		// STEP 1 - load raw image data, HOST->DEVICE, with/without pitch
		ImageInfo<DT> src;
		prepareData<false>("./res/terrain10x10.tif", src);

		// STEP 2 - create texture from the raw data
		//TextureInfo tiSrc = ... 
		TextureInfo tiSrc = createTextureObjectFrom2DArray(src);

		// STEP 3 - DO SOMETHING WITH THE TEXTURE
		dim3 block = { TPB_1D, TPB_1D,1 };
		dim3 grid{ (src.width + TPB_1D - 1) / TPB_1D, (src.height + TPB_1D - 1) / TPB_1D,  1 };
		float* dst = nullptr;
		hipMalloc((void**)&dst, src.width * src.height * sizeof(float));
		float gpuTime = GPUTIME(1, texKernel << <grid, block >> > (tiSrc.texObj, src.width, src.height, dst));
		printf("\x1B[93m[GPU time] %s: %f ms\033[0m\n", "getBest", gpuTime);
		checkDeviceMatrix<float>(dst, src.width * sizeof(float), src.height, src.width, "%6.1f ", "dst");

		// SETP 4 - release unused data
		//if (tiSrc.texObj) ... TODO;
		if (tiSrc.texArrayData) checkCudaErrors(hipDestroyTextureObject(tiSrc.texObj));
		//if (tiSrc.texArrayData) ... TODO;
		if (tiSrc.texArrayData) checkCudaErrors(hipFreeArray(tiSrc.texArrayData));
		if (src.dPtr) hipFree(src.dPtr);
		if (dst) hipFree(dst);

		hipDeviceSynchronize();
		//error = hipGetLastError();

		FreeImage_DeInitialise();
	}
} // namespace lab05