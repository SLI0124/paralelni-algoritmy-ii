#include <cudaDefs.h>
#include "lab01.h"
#include "lab02.h"
#include "lab03.h"
#include "lab04.h"

hipError_t error = hipSuccess;
hipDeviceProp_t deviceProp = hipDeviceProp_t();

int main(int argc, char** argv)
{
	initializeCUDA(deviceProp);
	//lab01::run();
	//lab02::run();
	//lab03::run();
	lab04::run();
	return 0;
}