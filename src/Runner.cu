#include <cudaDefs.h>
#include "lab01.h"
#include "lab02.h"
#include "lab03.h"

hipError_t error = hipSuccess;
hipDeviceProp_t deviceProp = hipDeviceProp_t();

int main(int argc, char** argv)
{
	initializeCUDA(deviceProp);
	//lab01::run();
	//lab02::run();
	lab03::run();
	return 0;
}