#include <cudaDefs.h>
#include "lab01.h"
#include "lab02.h"

hipError_t error = hipSuccess;
hipDeviceProp_t deviceProp = hipDeviceProp_t();


int main(int argc, char** argv)
{
	initializeCUDA(deviceProp);
	//lab01::run();
	lab02::run();
	return 0;
}