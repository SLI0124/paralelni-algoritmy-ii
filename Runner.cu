#include <cudaDefs.h>
#include "lab01.h"

hipError_t error = hipSuccess;
hipDeviceProp_t deviceProp = hipDeviceProp_t();


int main(int argc, char** argv)
{
	initializeCUDA(deviceProp);
	lab01::run();
	return 0;
}